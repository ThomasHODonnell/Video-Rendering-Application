#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void testKernel() {
	printf("%s", "Hello from the GPU!\n");
}

int main() {
	testKernel << <1, 1 >> > ();
	hipDeviceSynchronize();
	printf("%s", "Hello from the CPU!\n");
	return 0;
}