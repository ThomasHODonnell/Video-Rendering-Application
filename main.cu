#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include<glad/glad.h>
#include<GLFW/glfw3.h>

#include "Video.cuh"
#include "Frame.cuh"
#include "auxiliaryFunc.cuh"
#include "processingFunc.cuh"

using namespace cv;
using std::cout;
using std::endl;

const string path = "C:\\Users\\thoma\\source\\repos\\Video-Rendering-Application\\Videos\\Rubix.avi";


VideoCapture capture(path);
Mat frame;

__global__ void testK() {};

int main(int argc, char** argv) { 
	
	if (!capture.isOpened()) {
		throw invalid_argument("File upload failed.");
		return -1;
	}

	capture >> frame;

	Video V(frame);

	namedWindow("Video Input", 1);
	bool first = true;
    while (first) {

		capture >> frame;
		
        if (frame.empty())
            break;

		imshow("Video Input", frame);
		
		V.updateSequence(frame);
		Frame* current = new Frame(frame);

        // Press 'q' to exit the loop
        if (waitKey(30) >= 0)
            break;
		first = false;
    }
	waitKey(0);
	capture.release();
	destroyAllWindows();

	return 0;
}

